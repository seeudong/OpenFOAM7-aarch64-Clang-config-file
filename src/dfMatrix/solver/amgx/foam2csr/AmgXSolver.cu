#include "hip/hip_runtime.h"
/**
 * \file AmgXSolver.cpp
 * \brief Definition of member functions of the class AmgXSolver.
 * \author Pi-Yueh Chuang (pychuang@gwu.edu)
 * \author Matt Martineau (mmartineau@nvidia.com)
 * \date 2015-09-01
 * \copyright Copyright (c) 2015-2019 Pi-Yueh Chuang, Lorena A. Barba.
 * \copyright Copyright (c) 2019-2021, NVIDIA CORPORATION. All rights reserved.
 *            This project is released under MIT License.
 */

// AmgXWrapper
#include "AmgXSolver.H"
#include <numeric>
#include <limits>

// initialize AmgXSolver::count to 0
int AmgXSolver::count = 0;

// initialize AmgXSolver::rsrc to nullptr;
AMGX_resources_handle AmgXSolver::rsrc = nullptr;


/* \implements AmgXSolver::AmgXSolver */
AmgXSolver::AmgXSolver(const MPI_Comm &comm,
        const std::string &modeStr, const std::string &cfgFile)
{
    initialize(comm, modeStr, cfgFile);
}


/* \implements AmgXSolver::~AmgXSolver */
AmgXSolver::~AmgXSolver()
{
    if (isInitialised) finalize();
}


/* \implements AmgXSolver::initialize */
void AmgXSolver::initialize(const MPI_Comm &comm,
        const std::string &modeStr, const std::string &cfgFile)
{
    
    // if this instance has already been initialized, skip
    if (isInitialised) {
        fprintf(stderr,
                "This AmgXSolver instance has been initialized on this process.\n");
        exit(0);
    }

    // increase the number of AmgXSolver instances
    count += 1;

    // get the name of this node
    int     len;
    char    name[MPI_MAX_PROCESSOR_NAME];
    MPI_Get_processor_name(name, &len);  
    nodeName = name;

    // get the mode of AmgX solver
    setMode(modeStr);  

    // initialize communicators and corresponding information
    initMPIcomms(comm);  

    // only processes in gpuWorld are required to initialize AmgX
    if (gpuProc == 0)
    {
        initAmgX(cfgFile);  
    }

    // a bool indicating if this instance is initialized
    isInitialised = true;

    return;
}

void AmgXSolver::initialiseMatrixComms(
    AmgXCSRMatrix& matrix)
{
    matrix.initialiseComms(devWorld, gpuProc);
}

/* \implements AmgXSolver::setMode */
void AmgXSolver::setMode(const std::string &modeStr)
{
    if (modeStr == "dDDI")
        mode = AMGX_mode_dDDI;
    else if (modeStr == "dDFI")
        mode = AMGX_mode_dDFI;
    else if (modeStr == "dFFI")
        mode = AMGX_mode_dFFI;
    else if (modeStr[0] == 'h') {
        printf("CPU mode, %s, is not supported in this wrapper!",
                modeStr.c_str());
        exit(0);
    }
    else {
        printf("%s is not an available mode! Available modes are: "
                "dDDI, dDFI, dFFI.\n", modeStr.c_str());
        exit(0);
    }
}


/* \implements AmgXSolver::initAmgX */
 void AmgXSolver::initAmgX(const std::string &cfgFile)
{
    // only the first instance (AmgX solver) is in charge of initializing AmgX
    if (count == 1)
    {
        // initialize AmgX
        AMGX_SAFE_CALL(AMGX_initialize());

        // intialize AmgX plugings
        AMGX_SAFE_CALL(AMGX_initialize_plugins());

        // only the master process can output something on the screen
        // AMGX_SAFE_CALL(AMGX_register_print_callback(
        //             [](const char *msg, int length)->void
        //             {PetscPrintf(PETSC_COMM_WORLD, "%s", msg);}));

        // let AmgX to handle errors returned
        AMGX_SAFE_CALL(AMGX_install_signal_handler());
    }

    // create an AmgX configure object
    AMGX_SAFE_CALL(AMGX_config_create_from_file(&cfg, cfgFile.c_str()));

    // let AmgX handle returned error codes internally
    AMGX_SAFE_CALL(AMGX_config_add_parameters(&cfg, "exception_handling=1"));

    // create an AmgX resource object, only the first instance is in charge
    if (count == 1) AMGX_resources_create(&rsrc, cfg, &gpuWorld, 1, &devID);

    // create AmgX vector object for unknowns and RHS
    AMGX_vector_create(&AmgXP, rsrc, mode);
    AMGX_vector_create(&AmgXRHS, rsrc, mode);

    // create AmgX matrix object for unknowns and RHS
    AMGX_matrix_create(&AmgXA, rsrc, mode);

    // create an AmgX solver object
    AMGX_solver_create(&solver, rsrc, mode, cfg);

    // obtain the default number of rings based on current configuration
    AMGX_config_get_default_number_of_rings(cfg, &ring);
}

/* \implements AmgXSolver::finalize */
void AmgXSolver::finalize()
{
    // skip if this instance has not been initialised
    if (!isInitialised)
    {
        fprintf(stderr,
                "This AmgXWrapper has not been initialised. "
                "Please initialise it before finalization.\n");
        exit(0);
    }

    // only processes using GPU are required to destroy AmgX content
    if (gpuProc == 0)
    {
        // destroy solver instance
        AMGX_solver_destroy(solver);

        // destroy matrix instance
        AMGX_matrix_destroy(AmgXA);

        // destroy RHS and unknown vectors
        AMGX_vector_destroy(AmgXP);
        AMGX_vector_destroy(AmgXRHS);

        // only the last instance need to destroy resource and finalizing AmgX
        if (count == 1)
        {
            AMGX_resources_destroy(rsrc);
            AMGX_SAFE_CALL(AMGX_config_destroy(cfg));

            AMGX_SAFE_CALL(AMGX_finalize_plugins());
            AMGX_SAFE_CALL(AMGX_finalize());
        }
        else
        {
            AMGX_config_destroy(cfg);
        }

        // destroy gpuWorld
        MPI_Comm_free(&gpuWorld);  
    }

    // re-set necessary variables in case users want to reuse
    // the variable of this instance for a new instance
    gpuProc = MPI_UNDEFINED;
    MPI_Comm_free(&globalCpuWorld);
    MPI_Comm_free(&localCpuWorld);
    MPI_Comm_free(&devWorld);

    // decrease the number of instances
    count -= 1;

    // change status
    isInitialised = false;
}

/* \implements AmgXSolver::setOperator */
void AmgXSolver::setOperator
(
    const int nLocalRows,
    const int nGlobalRows,
    const int nLocalNz,
    AmgXCSRMatrix& matrix
)
{

    // Check the matrix size is not larger than tolerated by AmgX
    if(nGlobalRows > std::numeric_limits<int>::max())
    {
        fprintf(stderr,
                "AmgX does not support a global number of rows greater than "
                "what can be stored in 32 bits (nGlobalRows = %d).\n",
                nGlobalRows);
        exit(0);
    }

    const int nRows = (matrix.isConsolidated()) ? matrix.getNConsRows() : nLocalRows;
    const int nNz = (matrix.isConsolidated()) ? matrix.getNConsNz() : nLocalNz;

    if (nNz > std::numeric_limits<int>::max())
    {
        fprintf(stderr,
                "AmgX does not support non-zeros per (consolidated) rank greater than"
                "what can be stored in 32 bits (nLocalNz = %d).\n",
                nNz);
        exit(0);
    }

    // upload matrix A to AmgX
    if (gpuWorld != MPI_COMM_NULL)
    {
        MPI_Barrier(gpuWorld);  

        AMGX_distribution_handle dist;
        AMGX_distribution_create(&dist, cfg);

        // Must persist until after we call upload
        std::vector<int> offsets(gpuWorldSize + 1, 0);

        // Determine the number of rows per GPU
        std::vector<int> nRowsPerGPU(gpuWorldSize);
        MPI_Allgather(&nRows, 1, MPI_INT, nRowsPerGPU.data(), 1, MPI_INT, gpuWorld);  

        // Calculate the global offsets
        std::partial_sum(nRowsPerGPU.begin(), nRowsPerGPU.end(), offsets.begin() + 1);

        AMGX_distribution_set_partition_data(
            dist, AMGX_DIST_PARTITION_OFFSETS, offsets.data());

        // Set the column indices size, 32- / 64-bit
        AMGX_distribution_set_32bit_colindices(dist, true);

        AMGX_matrix_upload_distributed(
            AmgXA, nGlobalRows, nRows, nNz, 1, 1, matrix.getRowOffsets(),
            matrix.getColIndices(), matrix.getValues(), nullptr, dist);

        AMGX_distribution_destroy(dist);

        // bind the matrix A to the solver
        AMGX_solver_setup(solver, AmgXA);

        // connect (bind) vectors to the matrix
        AMGX_vector_bind(AmgXP, AmgXA);
        AMGX_vector_bind(AmgXRHS, AmgXA);
    }

    MPI_Barrier(globalCpuWorld);  
}


/* \implements AmgXSolver::updateOperator */
void AmgXSolver::updateOperator
(
    const int nLocalRows,
    const int nLocalNz,
    AmgXCSRMatrix& matrix
)
{
    const int nRows = (matrix.isConsolidated()) ? matrix.getNConsRows() : nLocalRows;
    const int nNz = (matrix.isConsolidated()) ? matrix.getNConsNz() : nLocalNz;

    // Replace the coefficients for the CSR matrix A within AmgX
    if (gpuWorld != MPI_COMM_NULL)
    {
        AMGX_matrix_replace_coefficients(AmgXA, nRows, nNz, matrix.getValues(), nullptr);

        // Re-setup the solver (a reduced overhead setup that accounts for consistent matrix structure)
        AMGX_solver_resetup(solver, AmgXA);
    }

    MPI_Barrier(globalCpuWorld);
}

/* \implements AmgXSolver::solve */
// void AmgXSolver::solve(
//     int nLocalRows, Vec& p, Vec& b, AmgXCSRMatrix& matrix)
// {
//     double* pscalar;
//     double* bscalar;

//     // get pointers to the raw data of local vectors
//     VecGetArray(p, &pscalar);
//     VecGetArray(b, &bscalar);

//     solve(nLocalRows, pscalar, bscalar, matrix);

//     VecRestoreArray(p, &pscalar);
//     VecRestoreArray(b, &bscalar);
// }


/* \implements AmgXSolver::solve */
void AmgXSolver::solve(
    int nLocalRows, double* pscalar, const double* bscalar, AmgXCSRMatrix& matrix)
{
    double* p;
    const double* b;
    int nRows;

    if (matrix.isConsolidated())
    {
        p = matrix.getPCons();
        b = matrix.getRHSCons();

        const int* rowDispls = matrix.getRowDispls();
        CHECK(hipMemcpy((void **)&p[rowDispls[myDevWorldRank]], pscalar, sizeof(double) * nLocalRows, hipMemcpyDefault));
        CHECK(hipMemcpy((void **)&b[rowDispls[myDevWorldRank]], bscalar, sizeof(double) * nLocalRows, hipMemcpyDefault));

        // Override the number of rows as the consolidated number of rows
        nRows = matrix.getNConsRows();

        // Sync as hipMemcpy to IPC buffers so device to device copies, which are non-blocking w.r.t host
        // All ranks in devWorld have the same value for isConsolidated
        CHECK(hipDeviceSynchronize());
        MPI_Barrier(devWorld);  
    }
    else
    {
        p = pscalar;
        b = bscalar;
        nRows = nLocalRows;
    }

    if (gpuWorld != MPI_COMM_NULL)
    {
        // Upload potentially consolidated vectors to AmgX
        AMGX_vector_upload(AmgXP, nRows, 1, p);
        AMGX_vector_upload(AmgXRHS, nRows, 1, b);

        MPI_Barrier(gpuWorld);  

        // Solve
        AMGX_solver_solve(solver, AmgXRHS, AmgXP);

        // Get the status of the solver
        AMGX_SOLVE_STATUS status;
        AMGX_solver_get_status(solver, &status);

        // Check whether the solver successfully solved the problem
        if (status != AMGX_SOLVE_SUCCESS)
        {
            fprintf(stderr, "AmgX solver failed to solve the system! "
                            "The error code is %d.\n",
                    status);
        }

        // Download data from device
        AMGX_vector_download(AmgXP, p);

        if(matrix.isConsolidated())
        {
            // AMGX_vector_download invokes a device to device copy, so it is essential that
            // the root rank blocks the host before other ranks copy from the consolidated solution
            CHECK(hipDeviceSynchronize());
        }
    }

    // If the matrix is consolidated, scatter the solution
    if (matrix.isConsolidated())
    {
        // Must synchronise before each rank attempts to read from the consolidated solution
        MPI_Barrier(devWorld);  

        const int* rowDispls = matrix.getRowDispls();

        // Ranks copy the portion of the solution they own into their rank-local buffers
        CHECK(hipMemcpy((void **)pscalar, &p[rowDispls[myDevWorldRank]], sizeof(double) * nLocalRows, hipMemcpyDefault));

        // Sync as hipMemcpy to IPC buffers so device to device copies, which are non-blocking w.r.t host
        // All ranks in devWorld have the same value for isConsolidated
        CHECK(hipDeviceSynchronize());
    }

    MPI_Barrier(globalCpuWorld);  
}


/* \implements AmgXSolver::getIters */
void AmgXSolver::getIters(int &iter)
{
    // only processes using AmgX will try to get # of iterations
    if (gpuProc == 0)
        AMGX_solver_get_iterations_number(solver, &iter);
}


/* \implements AmgXSolver::getResidual */
void AmgXSolver::getResidual(const int &iter, double &res)
{
    // only processes using AmgX will try to get residual
    if (gpuProc == 0)
        AMGX_solver_get_iteration_residual(solver, iter, 0, &res);
}

